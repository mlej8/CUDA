// https://developer.nvidia.com/blog/even-easier-introduction-cuda/

#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Function to add the elements of two arrays
 *        In order to turn the add function into a function that the GPU can run (called a kernel in CUDA)
 *        We need to add the specifier `__global__` to the function which tells the CUDA C++ compiler that this is a function that runs on the GPU and can be called from CPU code.
 *        Adding `__global__` transforms the function into a CUDA kernel function to add the elements of two arrays on the GPU
 *        These `__global__` functions are known as kernels, and code that runs on the GPU is often called device code, while code that runs on the CPU is host code.
 */
__global__ void
add(int n, float *x, float *y)
{
    /**
      * the following are variables that CUDA provides to let kernel get the indices of the running threads and thread blocks
      * threadIdx.x contains the index of the current thread within the block
      * blockDim.x contains the number of threads in the block
      * gridDim.x contains the number of blocks in the grid
      * blockIdx.x contains the index of the current thread block in the grid
      */

    // each thread gets its index by computing the offset to the beginning of its block + the thread's index within the
    // block (threadIdx.x)
    int index = blockDim.x * blockIdx.x + threadIdx.x; // idiomatic CUDA

    // in this case stride = total number of threads in the grid
    int stride = blockDim.x * gridDim.x;

    // this type of loop in a CUDA kernel is often called a `grid-stride loop`
    for (int i = index; i < n; i += stride) // NOTE: setting index to 0 and stride to 1 makes it semantically identical to the sequential version
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20; // 1M elements

    // allocate memory accessible by the GPU
    float *x, *y;

    /**
     *  UNIFIED memory https://developer.nvidia.com/blog/unified-memory-in-cuda-6/
     *  To compute on the GPU, we need to allocate memory accessible by the GPU. 
     *  Unified Memory in CUDA makes this easy by providing a single space accessible by all GPUs and CPUs in your system
     *  To allocate data in unified memory, we call 
     *
     *          cudaMallocManaged()
     *
     *  which returns a pointer that you can access from the host (CPU) code or device (GPU) code.
     *  To free the data, just pass the pointer to cudaFree().
     * 
     *  Hence, we need to replace the calls to `new` in the code with calls to cudaMallocManaged(), and replace calls to delete [] with calls to cudaFree.
     */

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /**
     * Launch the add() kernel, which invokes it on the GPU.
     * CUDA kernel launches are specified using the triple angle bracket syntax 
     * It represents the execution configuration and tells CUDA runtime how many parallel threads to use for the launch on the GPU
     * There are two parameters here:
     *  1. The first parameter of the execution configuration specifies the number of thread blocks.
     *  2. The second represents the number of threads in a thread block.
     * CUDA GPUs run kernels using blocks of threads that are a multiple of 32 in size, so 256 threads is a reasonable size to choose.
     * 
     *           <<<>>>
     * 
     * Using add<<<1, 256>>>(N, x, y);
     * __global__ void add(int n, float *x, float *y)
     *  {
     *      for (int i = 0; i < n; i++)
     *          y[i] = x[i] + y[i];
     *  }
     * 
     * will do the entire for loop computation once per thread, rather than spreading the computation across the parallel threads. 
     * We need to modify the kernel to spread the computation across parallel threads. 
     * CUDA C++ provides keywords that let kernels get the indices of the running threads. 
     * Specifically, threadIdx.x contains the index of the current thread within its block, and blockDim.x contains the number of threads in the block.
     * 
     * CUDA GPUs have many parallel processors grouped into Streaming Multiprocessors, or SMs. 
     * Each SM can run multiple concurrent thread blocks.
     * As an example, a Tesla P100 GPU based on the Pascal GPU Architecture has 56 SMs, each capable of supporting up to 2048 active threads. 
     * To take full advantage of all these threads, I should launch the kernel with multiple thread blocks.
     * 
     * The blocks of parallel threads make up what is known as the `grid`.
     * Since I have N elements to process and 256 threads per block, I just need to calculate the number of blocks to get at least N threads.
     * I simply divide N by the block size (being careful to round up in case N is not a multiple of blockSize).
     * 
     * 
     */
    int block_size = 256;
    int num_blocks = (N + block_size - 1) / block_size;
    add<<<num_blocks, block_size>>>(N, x, y);

    // we need to wait until the kernel is done before it accesses the results (because CUDA kernel launches don't block the calling CPU thread)
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}