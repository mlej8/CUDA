
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * @brief Function to add the elements of two arrays
 *        In order to turn the add function into a function that the GPU can run (called a kernel in CUDA)
 *        We need to add the specifier `__global__` to the function which tells the CUDA C++ compiler that this is a function that runs on the GPU and can be called from CPU code.
 *        Adding `__global__` transforms the function into a CUDA kernel function to add the elements of two arrays on the GPU
 *        These `__global__` functions are known as kernels, and code that runs on the GPU is often called device code, while code that runs on the CPU is host code.
 */
__global__ void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    // NOTE: setting index to 0 and stride to 1 makes it semantically identical to the sequential version
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20; // 1M elements

    // allocate memory accessible by the GPU
    float *x, *y;

    /**
     *  UNIFIED memory https://developer.nvidia.com/blog/unified-memory-in-cuda-6/
     *  To compute on the GPU, we need to allocate memory accessible by the GPU. 
     *  Unified Memory in CUDA makes this easy by providing a single space accessible by all GPUs and CPUs in your system
     *  To allocate data in unified memory, we call 
     *
     *          cudaMallocManaged()
     *
     *  which returns a pointer that you can access from the host (CPU) code or device (GPU) code.
     *  To free the data, just pass the pointer to cudaFree().
     * 
     *  Hence, we need to replace the calls to `new` in the code with calls to cudaMallocManaged(), and replace calls to delete [] with calls to cudaFree.
     */

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /**
     * Launch the add() kernel, which invokes it on the GPU.
     * CUDA kernel launches are specified using the triple angle bracket syntax 
     * It represents the execution configuration and tells CUDA runtime how many parallel threads to use for the launch on the GPU
     * There are two parameters here, the second is the number of threads in a thread block.
     * CUDA GPUs run kernels using blocks of threads that are a multiple of 32 in size, so 256 threads is a reasonable size to choose.
     * 
     *           <<<>>>
     * 
     * This launches one GPU thread to run add()
     * 
     * Using add<<<1, 256>>>(N, x, y);
     * __global__ void add(int n, float *x, float *y)
     *  {
     *      for (int i = 0; i < n; i++)
     *          y[i] = x[i] + y[i];
     *  }
     * 
     * will do the computation once per thread, rather than spreading the computation across the parallel threads. 
     * We need to modify the kernel to spread the computation across parallel threads. 
     * CUDA C++ provides keywords that let kernels get the indices of the running threads. 
     * Specifically, threadIdx.x contains the index of the current thread within its block, and blockDim.x contains the number of threads in the block.
     * 
     */
    add<<<1, 256>>>(N, x, y);

    // we need to wait until the kernel is done before it accesses the results (because CUDA kernel launches don't block the calling CPU thread)
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}