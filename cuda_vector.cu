#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>

__global__ void pass_vector(int *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] += 5;
    }
}

int main(void)
{
    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << std::endl;

    // print contents of H
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // resize H
    H.resize(2);

    std::cout << "H now has size " << H.size() << " and its content is:" << std::endl;
    // print contents of H
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << std::endl;

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    int n = 5;
    int *t;
    hipMallocManaged(&t, n * sizeof(int));

    std::cout << "before kernel: " << std::endl;
    for (int i = 0; i < n; i++)
    {
        t[i] = i;
        std::cout << t[i] << " " << std::endl;
    }
    pass_vector<<<1, 1>>>(t, n);
    hipDeviceSynchronize();
    std::cout << "after kernel: " << std::endl;
    for (int i = 0; i < n; i++)
    {
        std::cout << t[i] << " " << std::endl;
    }

    // print contents of D
    for (int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << std::endl;

    // H and D are automatically deleted when the function returns
    return 0;
}