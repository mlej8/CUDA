
#include <hip/hip_runtime.h>

/**
 * 1D Stencil
 *  Stencil codes are a class of iterative kernels which update array elements according to some fixed pattern, called a stencil.
 *  Consider applying a 1D stencil to a 1D array of elements.
 *  Each output element is the sum of input elements within a radius.
 * 
 *  If radius is 3, then each output element is the sum of 7 input elements:
 *  Input: [0,1,2,3,4,5,6]
 *          ^ ^ ^   ^ ^ ^
 *  Output: [21]
 * 
 * Implementation
 *  Read (ARRAY_SIZE + 2 * radius) input elements from global memory to shared memory 
 *  Compute ARRAY _SIZE output elements
 *  Write ARRAY _SIZE output elements to global memory
 */

__global__ void stencil() {
    // TODO
}


// TODO do one version with and without shared memory to compare
int main(int argc, char const *argv[])
{
    /* code */
    return 0;
}
