#include "hip/hip_runtime.h"
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/
#include <iostream>

/**
 * @brief Function to add the elements of two arrays
 *        In order to turn the add function into a function that the GPU can run (called a kernel in CUDA)
 *        We need to add the specifier `__global__` to the function which tells the CUDA C++ compiler that this is a function that runs on the GPU and can be called from CPU code.
 *        Adding `__global__` transforms the function into a CUDA kernel function to add the elements of two arrays on the GPU
 *        These `__global__` functions are known as kernels, and code that runs on the GPU is often called device code, while code that runs on the CPU is host code.
 */
__global__ void
add(int n, float *x, float *y)
{
    /**
      * the following are variables that CUDA provides to let kernel get the indices of the running threads and thread blocks
      * threadIdx.x contains the index of the current thread within the block
      * blockDim.x contains the number of threads in the block
      * gridDim.x contains the number of blocks in the grid
      * blockIdx.x contains the index of the current thread block in the grid
      */

    // each thread gets its index by computing the offset to the beginning of its block + the thread's index within the
    // block (threadIdx.x)
    int index = blockDim.x * blockIdx.x + threadIdx.x; // idiomatic CUDA

    // in this case stride = total number of threads in the grid
    int stride = blockDim.x * gridDim.x;

    // this type of loop in a CUDA kernel is often called a `grid-stride loop`
    for (int i = index; i < n; i += stride) // NOTE: setting index to 0 and stride to 1 makes it semantically identical to the sequential version
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20; // 1M elements

    // allocate memory accessible by the GPU
    float *x, *y;

    /**
     * UNIFIED MEMORY: https://developer.nvidia.com/blog/unified-memory-cuda-beginners/ and https://developer.nvidia.com/blog/unified-memory-in-cuda-6/
     * Unified Memory is a single memory address space accessible from any processor in a system.
     * To compute on the GPU, we need to allocate memory accessible by the GPU. 
     * Unified Memory in CUDA makes this easy by providing a single space accessible by all GPUs and CPUs in your system.
     * This hardware/software technology allows applications to allocate data that can be read or written from code running on either CPUs or GPUs.
     * Allocating Unified Memory is as simple as replacing calls to malloc() or `new` with calls to hipMallocManaged()
     * To allocate data in unified memory, we call 
     *
     *          hipMallocManaged()
     *
     * which returns a pointer that you can access from the host (CPU) code or device (GPU) code (any processor).
     * The memory pointed by this pointer is often called CUDA managed data. 
     * When code running on a CPU or GPU accesses data allocated this way, the CUDA system software and/or the hardware takes care of migrating memory pages to the memory of the accessing processor. 
     * The Pascal GPU architecture is the first with hardware support for ***virtual*** memory page faulting and migration, via its Page Migration Engine. 
     * Older GPUs based on the Kepler and Maxwell architectures also support a more limited form of Unified Memory.
     * 
     * To free the data, just pass the pointer to hipFree().
     * 
     * Hence, we need to replace the calls to `new` in the code with calls to hipMallocManaged(), and replace calls to delete [] with calls to hipFree.
     */

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    /**
     * Prefetching
     * 
     * Use Unified Memory prefetching to move the data to the GPU after initializing it to eliminate the migration overhead (from host to device) and get a more accurate measurement of the vector add kernel performance.
     * CUDA provides hipMemPrefetchAsync() for this purpose.
     * We can see that by doing this, there are no longer any GPU page faults reported.
     */
    // Prefetch the data to the GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N * sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N * sizeof(float), device, NULL);

    /**
     * Launch the add() kernel, which invokes it on the GPU.
     * CUDA kernel launches are specified using the triple angle bracket syntax 
     * It represents the execution configuration and tells CUDA runtime how many parallel threads to use for the launch on the GPU
     * There are two parameters here:
     *  1. The first parameter of the execution configuration specifies the number of thread blocks.
     *  2. The second represents the number of threads in a thread block.
     * CUDA GPUs run kernels using blocks of threads that are a multiple of 32 in size, so 256 threads is a reasonable size to choose.
     * 
     *           <<<>>>
     * 
     * Using add<<<1, 256>>>(N, x, y);
     * __global__ void add(int n, float *x, float *y)
     *  {
     *      for (int i = 0; i < n; i++)
     *          y[i] = x[i] + y[i];
     *  }
     * 
     * will do the entire for loop computation once per thread, rather than spreading the computation across the parallel threads. 
     * We need to modify the kernel to spread the computation across parallel threads. 
     * CUDA C++ provides keywords that let kernels get the indices of the running threads. 
     * Specifically, threadIdx.x contains the index of the current thread within its block, and blockDim.x contains the number of threads in the block.
     * 
     * CUDA GPUs have many parallel processors grouped into Streaming Multiprocessors, or SMs. 
     * Each SM can run multiple concurrent thread blocks.
     * As an example, a Tesla P100 GPU based on the Pascal GPU Architecture has 56 SMs, each capable of supporting up to 2048 active threads. 
     * To take full advantage of all these threads, I should launch the kernel with multiple thread blocks.
     * 
     * The blocks of parallel threads make up what is known as the `grid`.
     * Since I have N elements to process and 256 threads per block, I just need to calculate the number of blocks to get at least N threads.
     * I simply divide N by the block size (being careful to round up in case N is not a multiple of blockSize).
     */
    int block_size = 256;
    int num_blocks = (N + block_size - 1) / block_size;
    add<<<num_blocks, block_size>>>(N, x, y);

    // we need to wait until the kernel is done before it accesses the results (because CUDA kernel launches don't block the calling CPU thread)
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

/**
 * What Happens on Kepler When I call hipMallocManaged()?
 * 
 * On systems with pre-Pascal GPUs like the NVIDIA GeForce 600 and 700 series, calling hipMallocManaged() allocates `size` bytes of managed memory on the GPU device that is active when the call is made.
 * Internally, the driver also sets up page table entries in the GPU (device) memory for all pages covered by the allocation, so that the system knows that the pages are resident on that GPU.
 * 
 * So, in our example, running on a Tesla K80 GPU (Kepler architecture), x and y are both initially fully resident in GPU memory. 
 * Then in the loop starting on line 6, the CPU steps through both arrays, initializing their elements to 1.0f and 2.0f, respectively. 
 * Since the pages are initially resident in device memory, a page fault occurs on the CPU for each array page to which it writes, and the GPU driver migrates the page from device memory to CPU memory. 
 * After the loop, all pages of the two arrays are resident in CPU memory.
 * 
 * After initializing the data on the CPU, the program launches the add() kernel to add the elements of x to the elements of y: add<<<1, 256>>>(N, x, y);
 * 
 * On pre-Pascal GPUs, upon launching a kernel, the CUDA runtime must migrate all pages previously migrated to host memory or to another GPU back to the device memory of the device running the kernel. 
 * Since these older GPUs can’t page fault, all data must be resident on the GPU just in case the kernel accesses it (even if it won’t). 
 * This means there is potentially migration overhead on each kernel launch.
 */

/**
 * What Happens on Pascal When I call hipMallocManaged()?
 * 
 * On Pascal and later GPUs, managed memory may not be physically allocated when hipMallocManaged() returns; it may only be populated on access (or prefetching). 
 * In other words, pages and page table entries may not be created until they are accessed by the GPU or the CPU. 
 * The pages can migrate to any processor’s memory at any time, and the driver employs heuristics to maintain data locality and prevent excessive page faults. 
 * (Note: Applications can guide the driver using hipMemAdvise(), and explicitly migrate memory using hipMemPrefetchAsync(), as this blog post describes).
 * 
 * Unlike the pre-Pascal GPUs, the Tesla P100 supports hardware page faulting and migration. 
 * So in this case the runtime doesn’t automatically copy all the pages back to the GPU before running the kernel. 
 * The kernel launches without any migration overhead, and when it accesses any absent pages, the GPU stalls execution of the accessing threads, and the Page Migration Engine migrates the pages to the device before resuming the threads.
 * This means that the cost of the migrations is included in the kernel run time when I run my program on the Tesla P100 (2.1192 ms). 
 * 
 * In this kernel, every page in the arrays is written by the CPU, and then accessed by the CUDA kernel on the GPU, causing the kernel to wait on a lot of page migrations. 
 * That’s why the kernel time measured by the profiler is longer on a Pascal GPU like Tesla P100. 
 * Let’s look at the full nvprof output for the program on P100.
 * 
 * To eliminate/change the migration overhead to get a more accurate measurement of the vector add kernel performance:
 *  1. Move the data initialization to the GPU in another CUDA kernel.
 *  2. Run the kernel many times and look at the average and minimum run times. 
 *  3. Prefetch the data to GPU memory before running the kernel.
 * 
 */