
#include <hip/hip_runtime.h>
#include <iostream>

/**
 * Perform 2x2 max-pooling
 */
int main(int argc, char* argv[]) {
  if (argc != 3) {
    std::cout << "Usage: " << argv[0] <<" name of input png> <name of output png> <# threads>." << std::endl;
  }

  // split the image into 2x2 squares to determine number of threads needed
}
