#include "hip/hip_runtime.h"
#include <iostream>

#include "image_equality.hpp"
#include "lodepng.hpp"

using namespace std;

/**
 * @brief Perform 2x2 max-pooling.
 * // TODO generate description for arguments
 */
__global__ void pool(unsigned char* gpu_image, unsigned char* new_image, unsigned int width,
                     unsigned int height, unsigned int num_channels) {
    // TODO add check here if out of bound index then skip it.
    // TODO ask TA how to do the three dimensions with threads
    // TODO how do we do so that one thread is responsible for more than one block?
    // int index = threadIdx.x;  // index of current thread within block
    // int stride = blockDim.x;  // number of threads in a block
    // for (int i = (index * 2) % ; i < height; i += stride * 2) {
    for (int i = 0; i < height; i += 2) {
        // for (int j = (index * 2) % width; j < width; j += stride * 2) {
        for (int j = 0; j < width; j += 2) {
            for (int z = 0; z < num_channels; z++) {
                //  _________
                //  |   |   |
                //  |___|___|
                //  |   |   |
                //  |___|___|
                int flat_index = i * width * 4 + j * 4 + z;
                unsigned char values[4] = {gpu_image[flat_index],                               // top left
                                           gpu_image[flat_index + num_channels],                // top right
                                           gpu_image[flat_index + width * 4],                   // bottom left
                                           gpu_image[flat_index + width * 4 + num_channels]};   // bottom right
                unsigned char max_value = 0;
                for (int v = 0; v < 4; v++) {
                    if (values[v] > max_value) {
                        max_value = values[v];
                    }
                }
                
                new_image[(i / 2) * (width / 2) * 4 + (j / 2) * 4 + z] = max_value;
            }
        }
        // // one thread doing each of all three dimensions
        // int x = 0;
        // int y = 0;
        // int z = 0;
        // split the image into 2x2 squares to determine number of threads needed,
        // e.g. each thread will be responsible for one square
        // variables defined within device code do not need to be specified as device
        // variables because they are assumed to reside on the device.
    }
}

int main(int argc, char* argv[]) {
    // validate input parameters
    if (argc != 4) {
        std::cout << "Usage: " << argv[0]
                  << " <name of input png> <name of output png> <# threads>." << std::endl;
    }

    char* input_img_filename = argv[1];
    char* output_img_filename = argv[2];
    int num_threads = std::stoi(argv[3]);

    // 1. declare and allocate host and device memory
    unsigned char *image, *gpu_image, *new_image;
    unsigned int error, width, height, num_blocks, num_channels = 4;

    // 2. loading input image (initialize host data)
    error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    hipMalloc(&gpu_image, width * height * num_channels * sizeof(unsigned char));
    // new pooled image is going to be twice as small on each dimension - the
    // pooled image is accessible by both CPU and GPU
    hipMallocManaged(&new_image,
                      (width / 2) * (height / 2) * num_channels * sizeof(unsigned char));

    // 3. copy/transfer data from host to device
    hipMemcpy(gpu_image, image, width * height * num_channels * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // TODO optimize with number of blocks by computing number of total threads
    // needed to complete
    // num_blocks = (width / 2 * height / 2 * num_channels) / num_threads;
    num_blocks = 1;
    // TODO: what to do if we have more threads than needed - only use one thread
    // per 2x2 square
    // TODO: how can this be improved by leveraging all threads ? can we have more
    // than one thread per 2x2 cube
    // TODO: do we preserve the `a` dimension in rgba ?

    // execute kernels
    pool<<<num_blocks, num_threads>>>(gpu_image, new_image, width, height, num_channels);

    // tell CPU to wait until all threads in kernel are done execution before
    // accessing the resultsa
    hipDeviceSynchronize();

    // 5. Transfer results from device to host
    // hipMemcpy(image, gpu_image, width * height * num_channels *
    // sizeof(unsigned char), hipMemcpyDeviceToHost);

    // write pooled image
    error = lodepng_encode32_file(output_img_filename, new_image, (width / 2), (height / 2));
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    // clean memory
    hipFree(gpu_image);
    hipFree(new_image);
    free(image);
}
