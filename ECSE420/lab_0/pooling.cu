#include "hip/hip_runtime.h"
#include <iostream>

#include "lodepng.hpp"

using namespace std;

/**
 * @brief Perform 2x2 max-pooling.
 * 
 * @param gpu_image pointer to image on gpu memory.
 * @param new_image pointer to new image.
 * @param width width of image.
 * @param height height of image.
 * @param num_channels number of channels of the pooled image.
 */
__global__ void pool(unsigned char* gpu_image, unsigned char* new_image, unsigned int width,
                     unsigned int height, unsigned int num_channels) {
   
    // image index if it was 1D for a single channel
    int index = threadIdx.x * 2 + 2 * blockIdx.x * blockDim.x;
    int i = (index / width) * 2;
    int j = index % width;
    int z = blockIdx.y;
    int flat_index = i * width * 4 + j * 4 + z;
    unsigned char values[4] = {gpu_image[flat_index],                              // top left of 2x2 square
                               gpu_image[flat_index + num_channels],               // top right of 2x2 square
                               gpu_image[flat_index + width * 4],                  // bottom left of 2x2 square
                               gpu_image[flat_index + width * 4 + num_channels]};  // bottom right of 2x2 square
    unsigned char max_value = 0;
    for (int v = 0; v < 4; v++) {
        if (values[v] > max_value) {
            max_value = values[v];
        }
    }
    new_image[(i / 2) * (width / 2) * 4 + (j / 2) * 4 + z] = max_value;
}

int main(int argc, char* argv[]) {
    // validate input parameters
    if (argc != 4) {
        std::cout << "Usage: " << argv[0]
                  << " <name of input png> <name of output png> <# threads>." << std::endl;
        exit(1);
    }

    char* input_img_filename = argv[1];
    char* output_img_filename = argv[2];
    int block_size = std::stoi(argv[3]);

    // 1. declare and allocate host and device memory
    unsigned char *image, *gpu_image, *new_image;
    unsigned int error, width, height, num_channels = 4;

    // 2. loading input image (initialize host data)
    error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    hipMalloc(&gpu_image, width * height * num_channels * sizeof(unsigned char));
    
    // pooled image is going to be twice as small on each dimension 
    // using unified memory - pooled image is accessible by both CPU and GPU
    hipMallocManaged(&new_image,
                      (width / 2) * (height / 2) * num_channels * sizeof(unsigned char));

    // 3. copy/transfer data from host to device
    hipMemcpy(gpu_image, image, width * height * num_channels * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // rounding up in case image size is not a multiple of block_size
    dim3 num_blocks(((width / 2) * (height / 2) + block_size - 1) / block_size, num_channels, 1);

    // prefetch memory onto GPU
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(new_image, (width / 2) * (height / 2) * num_channels * sizeof(unsigned char), device, NULL);

    // execute kernels
    pool<<<num_blocks, block_size>>>(gpu_image, new_image, width, height, num_channels);

    // tell CPU to wait until all threads in kernel are done execution before
    // accessing the resultsa
    hipDeviceSynchronize();

    // 5. Transfer results from device to host
    // hipMemcpy(image, gpu_image, width * height * num_channels *
    // sizeof(unsigned char), hipMemcpyDeviceToHost);

    // write pooled image
    error = lodepng_encode32_file(output_img_filename, new_image, (width / 2), (height / 2));
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    // clean memory
    hipFree(gpu_image);
    hipFree(new_image);
    free(image);
}
