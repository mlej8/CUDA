#include "hip/hip_runtime.h"
#include <iostream>

#include "image_equality.hpp"
#include "lodepng.hpp"

using namespace std;

/**
 * @brief Perform 2x2 max-pooling.
 * In this exercise, do not worry about the case in which the image cannot be
 * divided perfectly into 2x2 squares – you may assume that the input test image
 * will have even width and height lengths
 */
__global__ void pool(unsigned char* gpu_image, unsigned char* new_image,
                     unsigned int width, unsigned int height) {
  // TODO add check here if out of bound index then skip it.
  // TODO ask TA how to do the three dimensions with threads
  // TODO how do we do so that one thread is responsible for more than one block?
  printf("hello");
  int index = threadIdx.x;  // index of current thread within block
  int stride = blockDim.x;  // number of threads in a block
  int num_channels = 4;
  for (int i = index * 2; i < height; i += stride * 2) {
    for (int j = index * 2; j < width; i += stride * 2)
      for (int z = 0; z < num_channels; z++) {
        //  _________
        //  |   |   |
        //  |___|___|
        //  |   |   |
        //  |___|___|
        int flat_index = i * width * 4 + j * 4;
        unsigned char values[4] = {gpu_image[flat_index + z],
                        gpu_image[flat_index + num_channels],
                        gpu_image[flat_index + z + width * 4],
                        gpu_image[flat_index + z + width * 4 + num_channels]};
        unsigned char max_value = 0;
        for (int v = 0; v < 4; v++) {
          if (values[v] > max_value) {
            max_value = values[v];
          }
        }
        new_image[(i/2) * (width/2) + (j/2) + z] = max_value;
      }
  }
  printf("done");
  // // one thread doing all three dimensions
  // int x = 0;
  // int y = 0;
  // int z = 0;
  // split the image into 2x2 squares to determine number of threads needed,
  // e.g. each thread will be responsible for one square
  // variables defined within device code do not need to be specified as device
  // variables because they are assumed to reside on the device.
}

int main(int argc, char* argv[]) {
  // validate input parameters
  if (argc != 4) {
    std::cout << "Usage: " << argv[0]
              << " <name of input png> <name of output png> <# threads>."
              << std::endl;
  }

  char* input_img_filename = argv[1];
  char* output_img_filename = argv[2];
  int num_threads = std::stoi(argv[3]);

  // 1. declare and allocate host and device memory
  unsigned char *image, *gpu_image, *new_image;
  unsigned int error, width, height, num_blocks, num_channels = 4;
  
  // 2. loading input image (initialize host data)
  error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
  if (error) {
    printf("Error %u: %s\n", error, lodepng_error_text(error));
    exit(error);
  }
  
  hipMalloc(&gpu_image, width * height * num_channels * sizeof(unsigned char));
  // new pooled image is going to be twice as small on each dimension - the
  // pooled image is accessible by both CPU and GPU
  hipMallocManaged(&new_image, (width / 2) * (height / 2) * num_channels *
                                    sizeof(unsigned char));

  // 3. copy/transfer data from host to device
  hipMemcpy(gpu_image, image,
             width * height * num_channels * sizeof(unsigned char),
             hipMemcpyHostToDevice);

  // TODO optimize with number of blocks by computing number of total threads
  // needed to complete
  // num_blocks = (width / 2 * height / 2 * num_channels) / num_threads;
  num_blocks = 1;
  // TODO: what to do if we have more threads than needed - only use one thread
  // per 2x2 square
  // TODO: how can this be improved by leveraging all threads ? can we have more
  // than one thread per 2x2 cube
  // TODO: do we preserve the `a` dimension in rgba ?

  // execute kernels
  pool<<<num_blocks, num_threads>>>(gpu_image, new_image, width, height);

  // tell CPU to wait until all threads in kernel are done execution before
  // accessing the results
  hipDeviceSynchronize();

  // 5. Transfer results from device to host
  // hipMemcpy(image, gpu_image, width * height * num_channels *
  // sizeof(unsigned char), hipMemcpyDeviceToHost);

  // write pooled image
  error = lodepng_encode32_file(output_img_filename, new_image, (width/2), (height/2));
  if (error) {
    printf("Error %u: %s\n", error, lodepng_error_text(error));
    exit(error);
  }

  // clean memory
  hipFree(gpu_image);
  hipFree(new_image);
  free(image);
}
