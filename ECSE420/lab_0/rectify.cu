
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>


__global__ void print_kernel() {
  printf("abc");
}

/**
 * Perform 2x2 max-pooling
 */
int main(int argc, char* argv[]) {
  print_kernel<<<1,1>>>();
  
  if (argc != 3) {
    std::cout << "Usage: " << argv[0] <<" name of input png> <name of output png> <# threads>." << std::endl;
  }

  // split the image into 2x2 squares to determine number of threads needed
}
