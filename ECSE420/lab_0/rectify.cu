#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "lodepng.hpp"

#define CENTER 127

/**
 * @brief Rectification produces an output image by repeating the following operation on each pixel of an input image.
 *        output[i][j] = input[i][j] if input[i][j] >= 0 else 0
 * 
 * @param image: image to be rectified.
 * @param num_channels: number of channels in the image to be rectified.
 */
__global__ void rectify(unsigned char* image, int num_channels) {
    int index = threadIdx.x * num_channels + blockDim.x * blockIdx.x * num_channels + blockIdx.y;
    if (image[index] < CENTER) image[index] = CENTER;
}

int main(int argc, char* argv[]) {
    // validate input parameters
    if (argc != 4) {
        std::cout << "Usage: " << argv[0]
                  << " <name of input png> <name of output png> <# threads>." << std::endl;
        exit(1);
    }

    char* input_img_filename = argv[1];
    char* output_img_filename = argv[2];
    int block_size = std::stoi(argv[3]);

    // 1. declare and allocate host and device memory
    unsigned char *image, *rectified_image;
    unsigned int error, width, height, num_channels = 4;

    // 2. loading input image (initialize host data)
    error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    hipMalloc(&rectified_image, width * height * num_channels * sizeof(unsigned char));

    // 3. copy/transfer data from host to device
    hipMemcpy(rectified_image, image, width * height * num_channels * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // rounding up in case image size is not a multiple of block_size
    dim3 num_blocks((width * height + (block_size - 1)) / block_size, num_channels, 1);

    // execute kernels
    rectify<<<num_blocks, block_size>>>(rectified_image, num_channels);

    // tell CPU to wait until all threads in kernel are done execution before
    // accessing the resultsa
    hipDeviceSynchronize();

    // 5. Transfer results from device to host
    hipMemcpy(image, rectified_image, width * height * num_channels *
    sizeof(unsigned char), hipMemcpyDeviceToHost);

    // write rectified image
    error = lodepng_encode32_file(output_img_filename, image, width, height);
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    // clean memory
    hipFree(rectified_image);
    free(image);
}
