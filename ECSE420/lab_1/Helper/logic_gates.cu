#include "hip/hip_runtime.h"
#include <stdio.h>
#include "logic_gates.cuh"

__global__ void logic_gate(const char *data, char *output, size_t input_file_length) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  
  if (index <  input_file_length) {
  int x = data[index * 3];
  int y = data[index * 3 + 1];
  int gate_type = data[index * 3 + 2];
  int result;
  switch (gate_type) {
    case AND:
      result = x & y;
      break;
    case OR:
      result = x | y;
      break;
    case NAND:
      result = !(x & y);
      break;
    case NOR:
      result = !(x + y);
      break;
    case XOR:
      result = x ^ y;
      break;
    case XNOR:
      result = x == y;
      break;
    default:
      printf("Error: Input gate '%d' invalid", gate_type);
      result = -1;
      break;
  }
  output[index] = result;
  }
}