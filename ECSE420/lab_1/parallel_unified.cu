#include <iostream>

#include "io.hpp"
#include "logic_gates.cuh"

using namespace std;

int main(int argc, char *argv[]) {
  char *input_file_path, *output_file_path;
  int input_file_length;
  if (argc != 4) {
    cout << "./parallel_unified <input_file_path> <input_file_length> <output_file_path>" << endl;
    exit(1);
  } else {
    input_file_path = argv[1];
    output_file_path = argv[3];
    input_file_length = atoi(argv[2]);
  }
  
  size_t data_size = input_file_length * 3 * sizeof(uint8_t);
  size_t output_size = input_file_length * sizeof(uint8_t);
  char *data, *output;
  hipMallocManaged(&data, data_size);
  hipMallocManaged(&output, output_size);
  ReadCSV(input_file_path, data);

  // prefetch memory onto GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(data, data_size, device, NULL);

  // assuming thhread size of 1024
  int num_blocks = 1, block_size = 1024;

  if (input_file_length > 1024) {
    num_blocks = (input_file_length + block_size - 1) / block_size;
  }
  // rounding up in case image size is not a multiple of block_size
  logic_gate<<<num_blocks, block_size>>>(data, output, input_file_length);
  hipDeviceSynchronize();
  
  // write final output
  WriteOutput(output_file_path, output, input_file_length);
  return 0;
}
