#include <iostream>

#include "io.hpp"
#include "logic_gates.cuh"

using namespace std;

int main(int argc, char *argv[]) {
  char *input_file_path, *output_file_path;
  int input_file_length;
  if (argc != 4) {
    cout << "./parallel_explicit <input_file_path> <input_file_length> "
            "<output_file_path>"
         << endl;
    exit(1);
  } else {
    input_file_path = argv[1];
    output_file_path = argv[3];
    input_file_length = stoi(argv[2]);
  }

  // 1. declare and allocate host and device memory
  size_t data_size = input_file_length * 3 * sizeof(uint8_t);
  size_t output_size = input_file_length * sizeof(uint8_t);
  char *data = new char[data_size];
  char *output = new char[output_size];
  char *gpu_data, *logic_gate_output;
  hipMalloc(&gpu_data, data_size);
  hipMalloc(&logic_gate_output, output_size);

  ReadCSV(input_file_path, data);

  // 2. copy/transfer data from host to device
  hipMemcpy(gpu_data, data, data_size, hipMemcpyHostToDevice);

  // assuming thhread size of 1024
  int num_blocks = 1, block_size = 1024;

  if (input_file_length > 1024) {
    num_blocks = (input_file_length + block_size - 1) / block_size;
  }
  // rounding up in case image size is not a multiple of block_size
  logic_gate<<<num_blocks, block_size>>>(gpu_data, logic_gate_output, input_file_length);
  hipDeviceSynchronize();

  // copy/transfer data from device to host
  hipMemcpy(output, logic_gate_output, output_size, hipMemcpyDeviceToHost);

  // write final output
  WriteOutput(output_file_path, output, input_file_length);
  return 0;
}
