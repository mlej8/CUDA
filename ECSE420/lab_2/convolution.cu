#include "hip/hip_runtime.h"
#include <iostream>

#include "lodepng.hpp"
#include "wm.h"

using namespace std;

/**
 * @brief Perform 2x2 max-pooling.
 *
 * @param gpu_image pointer to image on gpu memory.
 * @param new_image pointer to new image.
 * @param og_img_width width of original image.
 * @param og_img_height height of original image.
 * @param num_channels number of channels of original image.
 */
__global__ void convolve(unsigned char *gpu_image, unsigned char *new_image,
                         unsigned int og_img_width, unsigned int og_img_height,
                         unsigned int num_channels, float *filter) {
    // new image width
    int new_width = og_img_width - 2;

    // index in the new image
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // new image coordinates
    int i = index / new_width;  // integer division (discards any fractional remains)
    int j = index % new_width;
    int z = blockIdx.y;

    // original image coordinates
    int og_i = i + 1;
    int og_j = j + 1;

    // flat index for new image
    // int flat_index = i * new_width * (num_channels - 1) + j * (num_channels - 1) + z;
    if (z == 3) {  // not manipulating alpha channel (directly copying over)
        new_image[i * new_width * num_channels + j * num_channels + z] = gpu_image[og_i * og_img_width * 4 + og_j * 4 + z];
    } else {
        int sum =
            filter[0] * gpu_image[(og_i - 1) * og_img_width * 4 + (og_j - 1) * 4 + z] +
            filter[1] * gpu_image[(og_i - 1) * og_img_width * 4 + og_j * 4 + z] +
            filter[2] * gpu_image[(og_i - 1) * og_img_width * 4 + (og_j + 1) * 4 + z] +
            filter[3] * gpu_image[og_i * og_img_width * 4 + (og_j - 1) * 4 + z] +
            filter[4] * gpu_image[og_i * og_img_width * 4 + og_j * 4 + z] +
            filter[5] * gpu_image[og_i * og_img_width * 4 + (og_j + 1) * 4 + z] +
            filter[6] * gpu_image[(og_i + 1) * og_img_width * 4 + (og_j - 1) * 4 + z] +
            filter[7] * gpu_image[(og_i + 1) * og_img_width * 4 + og_j * 4 + z] +
            filter[8] * gpu_image[(og_i + 1) * og_img_width * 4 + (og_j + 1) * 4 + z];

        // clipping output
        if (sum > 255) {
            sum = 255;
        } else if (sum < 0) {
            sum = 0;
        }

        new_image[i * new_width * num_channels + j * num_channels + z] = sum;
    }
}

int main(int argc, char *argv[]) {
    char *input_img_filename, *output_img_filename;
    int block_size;
    if (argc != 4) {
        cout << "./convolve <name of input png> <name of output png> "
                "<# threads>"
             << endl;
        exit(1);
    } else {
        input_img_filename = argv[1];
        output_img_filename = argv[2];
        block_size = stoi(argv[3]);
    }
    // 1. declare and allocate host and device memory
    unsigned char *image, *gpu_image, *new_image_gpu, *new_image_cpu;
    float *filter;
    unsigned int error, width, height, num_channels = 4;
    // 2. loading input image (initialize host data)
    error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    size_t new_image_size = (width - 2) * (height - 2) * num_channels * sizeof(unsigned char);
    size_t image_size = width * height * num_channels * sizeof(unsigned char);

    hipMalloc(&gpu_image, image_size);
    hipMalloc(&filter, 3 * 3 * sizeof(float));  // size of filter

    // 3. copy/transfer data from host to device
    hipMemcpy(gpu_image, image,
               image_size,
               hipMemcpyHostToDevice);
    hipMemcpy(filter, w, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    // convolved image will be of size height − 2 by width − 2
    hipMalloc(&new_image_gpu, new_image_size);
    new_image_cpu = (unsigned char *) malloc(new_image_size);

    // rounding up in case image size is not a multiple of block_size
    dim3 num_blocks(((width - 2) * (height - 2) + block_size - 1) / block_size,
                    num_channels, 1);  // ignoring alpha channel

    // execute kernels
    convolve<<<num_blocks, block_size>>>(gpu_image, new_image_gpu, width, height,
                                         num_channels, filter);

    // tell CPU to wait until all threads in kernel are done execution before
    // accessing the results
    hipDeviceSynchronize();

    // 5. Transfer results from device to host
    hipMemcpy(new_image_cpu, new_image_gpu, new_image_size, hipMemcpyDeviceToHost);

    // write pooled image
    error = lodepng_encode32_file(output_img_filename, new_image_cpu, (width - 2),
                                  (height - 2));
    if (error) {
        printf("Error %u: %s\n", error, lodepng_error_text(error));
        exit(error);
    }

    // clean memory
    hipFree(gpu_image);
    hipFree(new_image_gpu);
    hipFree(filter);
    free(image);
    free(new_image_cpu);
}
