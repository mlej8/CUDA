#include "hip/hip_runtime.h"
#include <iostream>

#include "lodepng.hpp"
#include "wm.h"

using namespace std;

/**
 * @brief Perform 2x2 max-pooling.
 *
 * @param gpu_image pointer to image on gpu memory.
 * @param new_image pointer to new image.
 * @param og_img_width width of original image.
 * @param og_img_height height of original image.
 * @param num_channels number of channels of original image.
 */
__global__ void convolve(unsigned char *gpu_image, unsigned char *new_image,
                         unsigned int og_img_width, unsigned int og_img_height,
                         unsigned int num_channels, float *filter) {
  // new image width
  int new_width = og_img_width - 2;

  // index in the new image
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  // new image coordinates
  int i = index / new_width; // integer division (discards any fractional remains)
  int j = index % new_width;
  int z = blockIdx.y;

  // original image coordinates
  int og_i = i + 1;
  int og_j = j + 1;

  // flat index for new image
  // int flat_index = i * new_width * (num_channels - 1) + j * (num_channels - 1) + z;
  if (z == 3) { // not manipulating alpha channel (directly copying over)
    new_image[i * new_width * num_channels + j * num_channels + z] = gpu_image[og_i * og_img_width * 4 + og_j * 4 + z];
  } else {
  new_image[i * new_width * num_channels + j * num_channels + z] = 
      filter[0] * gpu_image[(og_i - 1) * og_img_width * 4 + (og_j - 1) * 4 + z] +
      filter[1] * gpu_image[(og_i - 1) * og_img_width * 4 + og_j * 4 + z] +
      filter[2] * gpu_image[(og_i - 1) * og_img_width * 4 + (og_j + 1) * 4 + z] +
      filter[3] * gpu_image[og_i * og_img_width * 4 + (og_j - 1) * 4 + z] +
      filter[4] * gpu_image[og_i * og_img_width * 4 + og_j * 4 + z] +
      filter[5] * gpu_image[og_i * og_img_width * 4 + (og_j + 1) * 4 + z] +
      filter[6] * gpu_image[(og_i + 1) * og_img_width * 4 + (og_j - 1) * 4 + z] +
      filter[7] * gpu_image[(og_i + 1) * og_img_width * 4 + og_j * 4 + z] +
      filter[8] * gpu_image[(og_i + 1) * og_img_width * 4 + (og_j + 1) * 4 + z];
  }

  // clipping output
  if (new_image[i * new_width * num_channels + j * num_channels + z] >  255) {
    new_image[i * new_width * num_channels + j * num_channels + z] = 255;
  } else if (new_image[i * new_width * num_channels + j * num_channels + z] < 0) {
    new_image[i * new_width * num_channels + j * num_channels + z] = 0;
  }
}

int main(int argc, char *argv[]) {
  char *input_img_filename, *output_img_filename;
  int block_size;
  if (argc != 4) {
    cout << "./convolve <name of input png> <name of output png> "
            "<# threads>"
         << endl;
    exit(1);
  } else {
    input_img_filename = argv[1];
    output_img_filename = argv[2];
    block_size = stoi(argv[3]);
  }
  // 1. declare and allocate host and device memory
  unsigned char *image, *gpu_image, *new_image;
  float *filter;
  unsigned int error, width, height, num_channels = 4;

  // 2. loading input image (initialize host data)
  error = lodepng_decode32_file(&image, &width, &height, input_img_filename);
  if (error) {
    printf("Error %u: %s\n", error, lodepng_error_text(error));
    exit(error);
  }

  hipMalloc(&gpu_image, width * height * num_channels * sizeof(unsigned char));
  hipMalloc(&filter, 3 * 3 * sizeof(float));  // size of filter

  // 3. copy/transfer data from host to device
  hipMemcpy(gpu_image, image,
             width * height * num_channels * sizeof(unsigned char),
             hipMemcpyHostToDevice);
  hipMemcpy(filter, w, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

  // convolved image will be of size height − 2 by width − 2
  // using unified memory - convolved image is accessible by both CPU and GPU
  hipMallocManaged(&new_image, (width - 2) * (height - 2) * num_channels *
                                    sizeof(unsigned char));

  // rounding up in case image size is not a multiple of block_size
  dim3 num_blocks(((width - 2) * (height - 2) + block_size - 1) / block_size,
                  num_channels, 1);  // ignoring alpha channel

  // prefetch memory onto GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(
      new_image,
      (width - 2) * (height - 2) * num_channels * sizeof(unsigned char), device,
      NULL);

  // execute kernels
  convolve<<<num_blocks, block_size>>>(gpu_image, new_image, width, height,
                                       num_channels, filter);

  // tell CPU to wait until all threads in kernel are done execution before
  // accessing the results
  hipDeviceSynchronize();

  // write pooled image
  error = lodepng_encode32_file(output_img_filename, new_image, (width - 2),
                                (height - 2));
  if (error) {
    printf("Error %u: %s\n", error, lodepng_error_text(error));
    exit(error);
  }

  // clean memory
  hipFree(gpu_image);
  hipFree(new_image);
  hipFree(filter);
  free(image);
}
