#include "hip/hip_runtime.h"
#include <iostream>
#include <unordered_set>

#include "io.hpp"
#include "logic_gates.cuh"
#include "read_input.hpp"

using namespace std;

__global__ void global_queuing(int numCurrLevelNodes,
                               int *currLevelNodes_h,
                               int *nodePtrs_h,
                               int *nodeNeighbors_h,
                               int *nodeVisited_h,
                               int *nodeOutput_h,
                               int *nodeGate_h,
                               int *nodeInput_h,
                               int *nextLevelNodes_h,
                               int *numNextLevelNodes_h) {
    int stride = blockDim.x * gridDim.x; // number of blocks * number of threads per block
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    //Loop over all nodes in the current level
    for (int i = index; i < numCurrLevelNodes; i += stride) {
        // extract element that is currently in the queue
        int node = currLevelNodes_h[i];

        // loop over all neighbors of the node
        for (int j = nodePtrs_h[node]; j < nodePtrs_h[node + 1]; j++) {
            int neighbor = nodeNeighbors_h[j];

            // if the neighbors hasn't been visited yet
            if (!nodeVisited_h[neighbor]) {
                // mark it as visited
                nodeVisited_h[neighbor] = 1;

                // update node output
                nodeOutput_h[neighbor] = logic_gate(nodeGate_h[neighbor], nodeOutput_h[node], nodeInput_h[neighbor]);

                // atomicAdd: reads a word at some address in global or shared memory, adds a number to it, and writes the result back to the same address. 
                // atomic means that it is guaranteed to be performed without interference from other threads. In other words, no other thread can access this address until the operation is complete.
                // add it to the global queue
                atomicExch(&nextLevelNodes_h[atomicAdd(numNextLevelNodes_h, 1)], neighbor);  // TODO find the index
            }
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc != 9) {
        cout << "Usage: ./global_queuing <path_to_input_1.raw> <path_to_input_2.raw> "
                "<path_to_input_3.raw> <path_to_input_4.raw> "
                "<output_nodeOutput_filepath> <output_nextLevelNodes_filepath> <numBlock> <blockSize>"
             << endl;
        exit(1);
    }
    char *input1 = argv[1];
    char *input2 = argv[2];
    char *input3 = argv[3];
    char *input4 = argv[4];
    char *output_nodeOutput_filepath = argv[5];
    char *output_nextLevelNodes_filepath = argv[6];
    int numBlock = atoi(argv[7]);
    int blockSize = atoi(argv[8]);

    unordered_set<int> valid_block_size{32, 64, 128};
    unordered_set<int> valid_num_blocks{10, 25, 35};
    if (valid_num_blocks.find(numBlock) == valid_num_blocks.end() || valid_block_size.find(blockSize) == valid_block_size.end()) {
        cout << "The valid block sizes are:\n";
        for (auto bs : valid_block_size) {
            cout << bs << "\n";
        }

        cout << "The valid number of blocks are:\n";
        for (auto block_num : valid_num_blocks) {
            cout << block_num << "\n";
        }
        exit(1);
    }

    // cpu variables
    int *nodePtrs_h;
    int *nodeNeighbors_h;
    int *nodeVisited_h;
    int *nodeGate_h;
    int *nodeInput_h;
    int *nodeOutput_h;
    int *currLevelNodes_h;

    // read input files
    int numNodePtrs = read_input_one_two_four(&nodePtrs_h, input1);
    int numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, input2);
    int numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h,
                                    &nodeOutput_h, input3);
    int numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input4);

    // same variables on the gpu
    int *gpu_nodePtrs_h;
    int *gpu_nodeNeighbors_h;
    int *gpu_nodeVisited_h;
    int *gpu_nodeGate_h;
    int *gpu_nodeInput_h;
    int *gpu_nodeOutput_h;
    int *gpu_currLevelNodes_h;

    // global queue will be using unified memory
    int *nextLevelNodes_h;
    int *numNextLevelNodes_h;

    // allocate memory on the GPU
    hipMalloc(&gpu_nodePtrs_h, numNodePtrs * sizeof(int));
    hipMalloc(&gpu_nodeNeighbors_h, numTotalNeighbors_h * sizeof(int));
    hipMalloc(&gpu_nodeVisited_h, numNodes * sizeof(int));
    hipMalloc(&gpu_nodeGate_h, numNodes * sizeof(int));
    hipMalloc(&gpu_nodeInput_h, numNodes * sizeof(int));
    hipMalloc(&gpu_nodeOutput_h, numNodes * sizeof(int));
    hipMalloc(&gpu_currLevelNodes_h, numCurrLevelNodes * sizeof(int));

    // copy content to GPU
    hipMemcpy(gpu_nodePtrs_h, nodePtrs_h, numNodePtrs * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_nodeNeighbors_h, nodeNeighbors_h, numTotalNeighbors_h * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_nodeVisited_h, nodeVisited_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_nodeGate_h, nodeGate_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_nodeInput_h, nodeInput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_nodeOutput_h, nodeOutput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_currLevelNodes_h, currLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);

    // using unified memory for global queue
    hipMallocManaged(&nextLevelNodes_h, numTotalNeighbors_h * sizeof(int));
    hipMallocManaged(&numNextLevelNodes_h, sizeof(int));

    // cout << *numNextLevelNodes_h;

    // launching kernel
    global_queuing<<<numBlock, blockSize>>>(
        numCurrLevelNodes,
        gpu_currLevelNodes_h,
        gpu_nodePtrs_h,
        gpu_nodeNeighbors_h,
        gpu_nodeVisited_h,
        gpu_nodeOutput_h,
        gpu_nodeGate_h,
        gpu_nodeInput_h,
        nextLevelNodes_h,
        numNextLevelNodes_h);
    hipDeviceSynchronize();

    // copy output back to CPU
    hipMemcpy(nodeOutput_h, gpu_nodeOutput_h, numNodes * sizeof(int), hipMemcpyDeviceToHost);

    WriteOutput(output_nodeOutput_filepath, nodeOutput_h, numNodes);
    WriteOutput(output_nextLevelNodes_filepath, nextLevelNodes_h, *numNextLevelNodes_h);
    hipFree(gpu_nodePtrs_h);
    hipFree(gpu_nodeNeighbors_h);
    hipFree(gpu_nodeVisited_h);
    hipFree(gpu_currLevelNodes_h);
    hipFree(gpu_nodeGate_h);
    hipFree(gpu_nodeInput_h);
    hipFree(gpu_nodeOutput_h);
    free(nodePtrs_h);
    free(nodeNeighbors_h);
    free(nodeVisited_h);
    free(currLevelNodes_h);
    free(nodeGate_h);
    free(nodeInput_h);
    free(nodeOutput_h);

    return 0;
}