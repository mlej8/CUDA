#include "hip/hip_runtime.h"
#include <iostream>
#include <unordered_set>

#include "io.hpp"
#include "logic_gates.cuh"
#include "read_input.hpp"

using namespace std;

__global__ void block_queuing(int numCurrLevelNodes, int blockQueueCapacity,
                              int *currLevelNodes_h, int *nodePtrs_h,
                              int *nodeNeighbors_h, int *nodeVisited_h,
                              int *nodeOutput_h, int *nodeGate_h,
                              int *nodeInput_h, int *nextLevelNodes_h,
                              int *numNextLevelNodes_h) {
  // initialize shared memory queue - extern indicates that __shared__ array
  // will be allocated dynamically at kernel launch time (shared memory size is
  // passed from the host)
  extern __shared__ int block_queue[];

  // counter used by threads within the same block
  __shared__ int blockQueueCounter;

  // use first thread in the block to initialize counter
  if (threadIdx.x == 0) {
    blockQueueCounter = 0;
  }
  __syncthreads();

  int stride =
      blockDim.x * gridDim.x;  // number of threads per block * number of blocks
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  // Loop over all nodes in the current level
  for (int i = index; i < numCurrLevelNodes; i += stride) {
    // extract element that is currently in the queue
    int node = currLevelNodes_h[i];

    // loop over all neighbors of the node
    for (int j = nodePtrs_h[node]; j < nodePtrs_h[node + 1]; j++) {
      int neighbor = nodeNeighbors_h[j];

      // if the neighbors hasn't been visited yet
      if (!nodeVisited_h[neighbor]) {
        // mark it as visited
        nodeVisited_h[neighbor] = 1;

        // update node output
        nodeOutput_h[neighbor] = logic_gate(
            nodeGate_h[neighbor], nodeOutput_h[node], nodeInput_h[neighbor]);

        /**
         * atomicAdd: reads a word at some address in global or shared memory,
         * adds a number to it, and writes the result back to the same address.
         * atomic means that it is guaranteed to be performed without
         * interference from other threads. In other words, no other thread can
         * access this address until the operation is complete.
         */

        if (blockQueueCounter <
            blockQueueCapacity) {  // add it to the block queue if there's still
                                   // space
          atomicExch(&block_queue[atomicAdd(&blockQueueCounter, 1)], neighbor);
        } else {  // queue is full, add it directly to the global queue
          atomicExch(&nextLevelNodes_h[atomicAdd(numNextLevelNodes_h, 1)],
                     neighbor);
          // TODO optimize to write shared memory to global memory once full
          // TODO allocate space for block queue to go into global queue
          // (instead of at the end...) -> restore blockQueueCounter
          // TODO store block queue in global queue
          // atomicExch(&blockQueueCounter, 0);
          // cudaMemSet(block_queue, )
        }
      }
    }
  }
  // we need to synchronize threads here, because we want to wait for the
  // blockQueue to be updated by all threads in the block, before continuing
  // (copy data from shared queue of the current block to the global queue)
  // (resource:
  // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#async_data_operations)
  __syncthreads();

  // add remaining data in the shared queue of the current block to the global
  // queue
  if (blockQueueCounter > 0) {
    for (int idx = threadIdx.x; idx < blockQueueCounter; idx += blockDim.x) {
      // copy data from shared memmory to global memory
      atomicExch(&nextLevelNodes_h[atomicAdd(numNextLevelNodes_h, 1)],
                 block_queue[idx]);
    }
  }
}

int main(int argc, char *argv[]) {
  if (argc != 10) {
    cout
        << "Usage: ./block_queuing <path_to_input_1.raw> <path_to_input_2.raw> "
           "<path_to_input_3.raw> <path_to_input_4.raw> "
           "<output_nodeOutput_filepath> <output_nextLevelNodes_filepath> "
           "<numBlock> <blockSize> <sharedQueueSize>"
        << endl;
    exit(1);
  }
  char *input1 = argv[1];
  char *input2 = argv[2];
  char *input3 = argv[3];
  char *input4 = argv[4];
  char *output_nodeOutput_filepath = argv[5];
  char *output_nextLevelNodes_filepath = argv[6];
  int numBlock = atoi(argv[7]);
  int blockSize = atoi(argv[8]);
  int blockQueueCapacity = atoi(argv[9]);

  unordered_set<int> valid_block_size{32, 64};
  unordered_set<int> valid_num_blocks{25, 35};
  unordered_set<int> valid_block_queue_capacity{32, 64};
  if (valid_num_blocks.find(numBlock) == valid_num_blocks.end() ||
      valid_block_size.find(blockSize) == valid_block_size.end() ||
      valid_block_queue_capacity.find(blockQueueCapacity) ==
          valid_block_queue_capacity.end()) {
    cout << "The valid block sizes are:\n";
    for (auto bs : valid_block_size) {
      cout << bs << "\n";
    }

    cout << "The valid number of blocks are:\n";
    for (auto block_num : valid_num_blocks) {
      cout << block_num << "\n";
    }

    cout << "The valid number of block queue capacity are:\n";
    for (auto capacity : valid_block_queue_capacity) {
      cout << capacity << "\n";
    }
    exit(1);
  }

  // cpu variables
  int *nodePtrs_h;
  int *nodeNeighbors_h;
  int *nodeVisited_h;
  int *nodeGate_h;
  int *nodeInput_h;
  int *nodeOutput_h;
  int *currLevelNodes_h;

  // read input files
  int numNodePtrs = read_input_one_two_four(&nodePtrs_h, input1);
  int numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, input2);
  int numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h,
                                  &nodeOutput_h, input3);
  int numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input4);

  // same variables on the gpu
  int *gpu_nodePtrs_h;
  int *gpu_nodeNeighbors_h;
  int *gpu_nodeVisited_h;
  int *gpu_nodeGate_h;
  int *gpu_nodeInput_h;
  int *gpu_nodeOutput_h;
  int *gpu_currLevelNodes_h;

  // global queue will be using unified memory
  int *nextLevelNodes_h;
  int *numNextLevelNodes_h;

  // allocate memory on the GPU
  hipMalloc(&gpu_nodePtrs_h, numNodePtrs * sizeof(int));
  hipMalloc(&gpu_nodeNeighbors_h, numTotalNeighbors_h * sizeof(int));
  hipMalloc(&gpu_nodeVisited_h, numNodes * sizeof(int));
  hipMalloc(&gpu_nodeGate_h, numNodes * sizeof(int));
  hipMalloc(&gpu_nodeInput_h, numNodes * sizeof(int));
  hipMalloc(&gpu_nodeOutput_h, numNodes * sizeof(int));
  hipMalloc(&gpu_currLevelNodes_h, numCurrLevelNodes * sizeof(int));

  // copy content to GPU
  hipMemcpy(gpu_nodePtrs_h, nodePtrs_h, numNodePtrs * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_nodeNeighbors_h, nodeNeighbors_h,
             numTotalNeighbors_h * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_nodeVisited_h, nodeVisited_h, numNodes * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_nodeGate_h, nodeGate_h, numNodes * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_nodeInput_h, nodeInput_h, numNodes * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_nodeOutput_h, nodeOutput_h, numNodes * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_currLevelNodes_h, currLevelNodes_h,
             numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);

  // using unified memory for global queue
  hipMallocManaged(&nextLevelNodes_h, numTotalNeighbors_h * sizeof(int));
  hipMallocManaged(&numNextLevelNodes_h, sizeof(int));

  // launching kernel
  block_queuing<<<numBlock, blockSize,
                  blockQueueCapacity * sizeof(int)>>>(  // pass the size of the
                                                        // per-block shared
                                                        // queue (__shared__
                                                        // block_queue)
      numCurrLevelNodes, blockQueueCapacity, gpu_currLevelNodes_h,
      gpu_nodePtrs_h, gpu_nodeNeighbors_h, gpu_nodeVisited_h, gpu_nodeOutput_h,
      gpu_nodeGate_h, gpu_nodeInput_h, nextLevelNodes_h, numNextLevelNodes_h);
  hipDeviceSynchronize();

  // copy output back to CPU
  hipMemcpy(nodeOutput_h, gpu_nodeOutput_h, numNodes * sizeof(int),
             hipMemcpyDeviceToHost);

  WriteOutput(output_nodeOutput_filepath, nodeOutput_h, numNodes);
  WriteOutput(output_nextLevelNodes_filepath, nextLevelNodes_h,
              *numNextLevelNodes_h);
  hipFree(gpu_nodePtrs_h);
  hipFree(gpu_nodeNeighbors_h);
  hipFree(gpu_nodeVisited_h);
  hipFree(gpu_currLevelNodes_h);
  hipFree(gpu_nodeGate_h);
  hipFree(gpu_nodeInput_h);
  hipFree(gpu_nodeOutput_h);
  hipFree(nextLevelNodes_h);
  hipFree(numNextLevelNodes_h);
  free(nodePtrs_h);
  free(nodeNeighbors_h);
  free(nodeVisited_h);
  free(currLevelNodes_h);
  free(nodeGate_h);
  free(nodeInput_h);
  free(nodeOutput_h);

  return 0;
}
